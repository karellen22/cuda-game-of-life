#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>
#include <sstream>
#include "Game.h"
#include <chrono>
#include <iostream>

#define I 25
#define J 25

bool cells[I][J];
__device__ bool dev_cells[I][J];

void fillArray(std::vector<std::vector<bool>> cellsVector) 
{
    for (int i = 0; i < I; i++) {
        int j = 0;
        for (
            auto it = cellsVector[i].begin();
            it != cellsVector[i].end(); it++)
        {
            cells[i][j] = *it == 1;
            ++j;
        }
        std::cout << std::endl;
    }
}

void playCpu(Game &game) 
{
    auto start = std::chrono::high_resolution_clock::now();
    game.PlayGame();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    auto durationMilisec = duration / (double)1000000;
    std::cout << "CPU running time on "<< I << "x" << J << " board: "  <<durationMilisec << " s" <<std::endl;
}

__global__ void playGpu()
{
    __shared__ bool shr_cells[I][J];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto fakeJ = 0;
    auto fakeI = 0;
    if (idx >= J) {
        fakeI = idx / J;
        fakeJ = idx - ( fakeI * J );
    }
    else {
        fakeJ = idx;
    }
    shr_cells[fakeI][fakeJ] = dev_cells[fakeI][fakeJ];
    __syncthreads();

    auto count = 0;

    if (fakeI > 0 && fakeJ > 0) {
        if (shr_cells[fakeI - 1][fakeJ - 1]) ++count;
    }
    // top
    if (fakeI > 0) {
        if (shr_cells[fakeI - 1][fakeJ]) ++count;
    }
    // top right
    if (fakeI > 0 && fakeJ < J - 1) {
        if (shr_cells[fakeI - 1][fakeJ + 1]) ++count;
    }
    // right
    if (fakeJ < J - 1) {
        if (shr_cells[fakeI][fakeJ + 1]) ++count;
    }
    // bottom right
    if (fakeI < I - 1 && fakeJ < J - 1) {
        if (shr_cells[fakeI + 1][fakeJ + 1]) ++count;
    }
    // bottom
    if (fakeI < I - 1) {
        if (shr_cells[fakeI + 1][fakeJ]) ++count;
    }
    // bottom left
    if (fakeI < I - 1 && fakeJ > 0) {
        if (shr_cells[fakeI + 1][fakeJ - 1]) ++count;
    }
    // left
    if (fakeJ > 0) {
        if (shr_cells[fakeI][fakeJ - 1]) ++count;
    }

    __syncthreads();

    if (shr_cells[fakeI][fakeJ] == true)
    {
        if (!(count == 2 || count == 3))
        {
        shr_cells[fakeI][fakeJ] = false;
        }
    }
    else
    {
        if (count == 3)
        {
        shr_cells[fakeI][fakeJ] = true;
        }
    }
    dev_cells[fakeI][fakeJ] = shr_cells[fakeI][fakeJ];

    __syncthreads();
}

void displayBoard()
{
    for (int i = 0; i < I; i++) {
        for (int j = 0; j < J; j++)
        {
            auto displayVariable = cells[i][j] == 1 ? "+" : "-";
            //auto displayVariable = *it == 1 ? getNeighboursAlive(i, j) : 0;
            //std::cout << displayVariable;
            std::cout << displayVariable << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main()
{
    // sor, oszlop
    auto game1 = Game(I, J);
    auto gameTable = game1.m_Cells;
    //playCpu(game1);
    fillArray(gameTable);
    displayBoard();
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockNumber = 1;
    int threadNumber = I*J;
    if (threadNumber > 1024)
    {
        blockNumber = threadNumber % 1024 == 0 ? threadNumber / 1024 : threadNumber / 1024 + 1;
        threadNumber = 1024;
    }


    hipEventRecord(start);
    for (size_t i = 0; i < 100; i++)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(dev_cells), cells, I * J * sizeof(bool));
        playGpu << < blockNumber, threadNumber >> > ();
        hipMemcpyFromSymbol(cells, HIP_SYMBOL(dev_cells), I * J * sizeof(bool));
    }
    
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU running time: " << milliseconds << " ms" << std::endl;
    displayBoard();

    return 0;
}